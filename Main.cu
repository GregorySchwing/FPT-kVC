#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "cuda_by_example/common/book.h"


__global__ void kernel(void){

}

__global__ void add(int a, int b, int *c){
    *c = a + b;
}

int main(int argc, char *argv[])
{
    int c;
    int *dev_c;
    HANDLE_ERROR( hipMalloc((void**)&dev_c, sizeof(int)));

    add<<<1,1>>>( 2, 7, dev_c);

    HANDLE_ERROR( hipMemcpy(   &c, 
                                dev_c,
                                sizeof(int),
                                hipMemcpyDeviceToHost
                            ));

    printf( "2 + 7 = %d\n", c);
    hipFree(dev_c);

    //kernel<<<1,1>>>();
    //printf("Hello, World!\n");
    return 0;

}