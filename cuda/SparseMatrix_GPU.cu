
#include <hip/hip_runtime.h>
#ifdef FPT_CUDA

#include "SparseMatrix_GPU.cuh"
#include <memory>

/* Copy constr */
__host__ __device__ SparseMatrix_GPU::SparseMatrix_GPU(const SparseMatrix & s):
size(s.size), 
numberOfRows(s.numberOfRows), 
numberOfColumns(s.numberOfColumns){
//    std::cout << "Setting size, numRows, numCols - Reserving new_vals" << std::endl;
    // A copy for writing purposes
    //cudaMalloc(&new_values_dev, s.size*sizeof(int)); 
}

__host__ __device__ SparseMatrix_GPU::~SparseMatrix_GPU(){
    //cudaFree(new_values_dev);
}

#endif