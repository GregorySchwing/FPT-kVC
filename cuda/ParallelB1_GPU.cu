
#include <hip/hip_runtime.h>
#ifdef FPT_CUDA

#include "ParallelB1_GPU.cuh"
#include <math.h>       /* pow */
#include "cub/cub.cuh"

// Sum of i = 0 to n/2
// 3^i

__device__ int randomGPU(unsigned int counter, ulong step, ulong seed)
{
  RNG::ctr_type c = {{}};
  RNG::ukey_type uk = {{}};
  uk[0] = step;
  uk[1] = seed;
  RNG::key_type k = uk;
  c[0] = counter;
  RNG::ctr_type r = philox4x32(c, k);
  return r[0];
}

__device__ RNG::ctr_type randomGPU_four(unsigned int counter, ulong step, ulong seed)
{
  RNG::ctr_type c = {{}};
  RNG::ukey_type uk = {{}};
  uk[0] = step;
  uk[1] = seed;
  RNG::key_type k = uk;
  c[0] = counter;
  RNG::ctr_type r = philox4x32(c, k);
  return r;
}

__host__ __device__ int CalculateWorstCaseSpaceComplexity(int vertexCount){
    int summand= 0;
    // ceiling(vertexCount/2) loops
    for (int i = 0; i < (vertexCount + 2 - 1)/2; ++i)
        summand += pow (3.0, i);
    return summand;
}

__host__ __device__ long long CalculateSpaceForDesiredNumberOfLevels(int NumberOfLevels){
    long long summand= 0;
    // ceiling(vertexCount/2) loops
    for (int i = 0; i < NumberOfLevels; ++i)
        summand += pow (3.0, i);
    return summand;
}

__host__ __device__ long long CalculateDeepestLevelWidth(int deepestLevelSize){
    long long summand= 0;
    summand += pow (3.0, deepestLevelSize);
    return summand;
}


__host__ __device__ long long CalculateSizeRequirement(int startingLevel,
                                                        int endingLevel){
    long long summand= 0;
    // ceiling(vertexCount/2) loops
    for (int i = startingLevel; i < endingLevel; ++i)
        summand += pow (3.0, i);
    return summand;
}

__host__ __device__ long long CalculateLevelOffset(int level){
    if(level == 0)
        return 0;
    else
        return pow(3.0, (level-1)) + 1;
}

__host__ __device__ long long CalculateLevelUpperBound(int level){
    if(level == 0)
        return 1;
    else
        return pow(3.0, (level)) + 1;
}

typedef int inner_array_t[2];

__global__ void InduceSubgraph( int numberOfRows,
                                int edgesLeftToCover,
                                int * old_row_offsets_dev,
                                int * old_columns_dev,
                                int * old_values_dev,
                                int * new_row_offsets_dev,
                                int * new_columns_dev,
                                int * new_values_dev){

    //int row = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.x;

    inner_array_t *C_ref = new inner_array_t[numberOfRows];

    for (int iter = row; iter < numberOfRows; iter += blockDim.x){

        //printf("Thread %d, row %d", threadIdx.x, iter);
        C_ref[iter][0] = 0;
        C_ref[iter][1] = 0;
        //printf("Thread %d, row %d, old_row_offsets_dev[iter] = %d", threadIdx.x, iter, old_row_offsets_dev[iter]);
        //printf("Thread %d, row %d, old_row_offsets_dev[iter+1] = %d", threadIdx.x, iter, old_row_offsets_dev[iter+1]);
        //printf("Thread %d, row %d, old_values_dev[endOffset] = %d", threadIdx.x, iter, old_values_dev[old_row_offsets_dev[iter+1]]);

        int beginIndex = old_row_offsets_dev[iter];
        int endIndex = old_row_offsets_dev[iter+1];

        for (int i = beginIndex; i < endIndex; ++i){
            ++C_ref[iter][old_values_dev[i]];
        }

        // This is  [old degree - new degree , new degree]
        for (int i = 1; i < 2; ++i){
            C_ref[iter][i] = C_ref[iter][i] + C_ref[iter][i-1];
        }
        //printf("Thread %d, row %d, almost done", threadIdx.x, iter);

        /* C_ref[A_row_indices[i]]]-1 , because the values of C_ref are from [1, n] -> [0,n) */
        for (int i = endIndex-1; i >= beginIndex; --i){
            if (old_values_dev[i]){
                new_columns_dev[new_row_offsets_dev[iter] - C_ref[iter][0] + C_ref[iter][1]-1] = old_columns_dev[i];
                new_values_dev[new_row_offsets_dev[iter] - C_ref[iter][0] + C_ref[iter][1]-1] = old_values_dev[i];
                --C_ref[iter][old_values_dev[i]];
            }
        }
        if (row == 0){
            printf("Block %d induced root of graph", blockIdx.x);
            for (int i = 0; i < edgesLeftToCover; ++i){
                printf("%d ",new_columns_dev[i]);
            }
            printf("\n");
            for (int i = 0; i < edgesLeftToCover; ++i){
                printf("%d ",new_values_dev[i]);
            }
            printf("\n");
        }
    }
    delete[] C_ref;
}

__global__ void SetEdges(int numberOfRows,
                        int numberOfEdgesPerGraph,
                        int levelOffset,
                        int levelUpperBound,
                        int * global_row_offsets_dev_ptr,
                        int * global_columns_dev_ptr,
                        int * global_values_dev_ptr,
                        int * global_paths_ptr,
                        int * global_paths_length,
                        int * global_edges_left_to_cover_count){

    int leafIndex = levelOffset + blockIdx.x;
    if (leafIndex >= levelUpperBound) return;

    int threadIndex = threadIdx.x;

    int rowOffsOffset = (numberOfRows + 1) * (leafIndex-1)/3;
    int valsAndColsOffset = numberOfEdgesPerGraph * leafIndex;
    if (global_row_offsets_dev_ptr[rowOffsOffset + numberOfRows -1] != global_edges_left_to_cover_count[(leafIndex-1)/3])
        printf("\n\n\nERROR in the row offsets (%d) and number of columns (%d)\n\n\n", 
            global_row_offsets_dev_ptr[rowOffsOffset + numberOfRows -1],
            global_edges_left_to_cover_count[(leafIndex-1)/3]);
    int children[2], LB, UB, v, vLB, vUB;
    // Parent's DFS path
    int pathsOffset = ((leafIndex-1)/3) * 4;
/*
child x (path[0]) (path[2]);

        (path[1]) (path[3]);     

child y         or

        (path[1]) (path[0]);    

child z (path[2]) (path[1]);

Can't figure out a way to avoid these if conditionals without a kernel call to classify before this kernel is called.
*/
    int pathType = leafIndex % 3;
    if (pathType == 0){
        children[0] = global_paths_ptr[pathsOffset];
        children[1] = global_paths_ptr[pathsOffset + 2];
    } else if (pathType == 1) { 
        children[0] = global_paths_ptr[pathsOffset + 1];
        children[1] = global_paths_ptr[pathsOffset + 2];
    } else {
        children[0] = global_paths_ptr[pathsOffset + 1];
        if (global_paths_ptr[pathsOffset] == global_paths_ptr[pathsOffset + 3])
            children[1] = global_paths_ptr[pathsOffset];
        else
            children[1] = global_paths_ptr[pathsOffset + 3];
    }
    // Set out-edges
    for (int i = 0; i < 2; ++i){
        LB = global_row_offsets_dev_ptr[rowOffsOffset + children[i]];
        UB = global_row_offsets_dev_ptr[rowOffsOffset + children[i] + 1];    
        for (int edge = LB + threadIndex; edge < UB; edge += blockDim.x){
            global_values_dev_ptr[valsAndColsOffset + edge] = 0;
        }
    }
    __syncthreads();
    if (threadIndex == 0 && blockIdx.x == 0){
        printf("Block %d, levelOffset %d, leafIndex %d, children removed %d %d\n", blockIdx.x, levelOffset, leafIndex, children[0], children[1]);
        for (int i = 0; i < global_edges_left_to_cover_count[(leafIndex-1)/3]; ++i){
            printf("(%d, %d) ",global_columns_dev_ptr[valsAndColsOffset + i], global_values_dev_ptr[valsAndColsOffset + i]);
        }
        printf("\n");
    }
    // (u,v) is the form of edge pairs.  We are traversing over v's outgoing edges, 
    // looking for u as the destination and turning off that edge.
    // this may be more elegantly handled by 
    // (1) an associative data structure
    // (2) an undirected graph 
    // Parallel implementations of both of these need to be investigated.
    for (int i = 0; i < 2; ++i){
        LB = global_row_offsets_dev_ptr[rowOffsOffset + children[i]];
        UB = global_row_offsets_dev_ptr[rowOffsOffset + children[i] + 1];    // Set out-edges
        for (int edge = LB + threadIndex; edge < UB; edge += blockDim.x){
            v = global_columns_dev_ptr[valsAndColsOffset + edge];
            // guarunteed to only have one incoming and one outgoing edge connecting (x,y)
            vLB = global_row_offsets_dev_ptr[rowOffsOffset + v];
            vUB = global_row_offsets_dev_ptr[rowOffsOffset + v + 1];
            for (int outgoingEdgeOfV = vLB + threadIndex; 
                    outgoingEdgeOfV < vUB; 
                        outgoingEdgeOfV += blockDim.x){
                if (children[i] == global_columns_dev_ptr[valsAndColsOffset + outgoingEdgeOfV]){
                    // Set in-edge
                    global_values_dev_ptr[valsAndColsOffset + outgoingEdgeOfV] = 0;
                }
            }
        }
    }
    __syncthreads();
    if (threadIndex == 0 && blockIdx.x == 0){
        printf("Block %d, levelOffset %d, leafIndex %d, children removed %d %d\n", blockIdx.x, levelOffset, leafIndex, children[0], children[1]);
        for (int i = 0; i < global_edges_left_to_cover_count[(leafIndex-1)/3]; ++i){
            printf("(%d, %d) ",global_columns_dev_ptr[valsAndColsOffset + i], global_values_dev_ptr[valsAndColsOffset + i]);
        }
        printf("\n");
    }
}

// Sets the new degrees without the edges and the edges left to cover
__global__ void SetDegreesAndCountEdgesLeftToCover(int numberOfRows,
                            int numberOfEdgesPerGraph,
                            int levelOffset,
                            int levelUpperBound,
                            int * global_row_offsets_dev_ptr,
                            int * global_values_dev_ptr,
                            int * global_degrees_dev_ptr,
                            int * global_edges_left_to_cover_count){

    int leafIndex = levelOffset + blockIdx.x;
    if (leafIndex >= levelUpperBound) return;

    extern __shared__ int degrees[];

    // Use parent's row offs
    int rowOffsOffset = (numberOfRows + 1) * (leafIndex-1)/3;
    // my vals some of which are now 0
    int valsAndColsOffset = numberOfEdgesPerGraph * leafIndex;
    int degreesOffset = leafIndex * numberOfRows;
    int LB, UB, iter, row, edge;

    row = threadIdx.x;
    for (iter = row; iter < numberOfRows; iter += blockDim.x){
        LB = global_row_offsets_dev_ptr[rowOffsOffset + iter];
        UB = global_row_offsets_dev_ptr[rowOffsOffset + iter + 1];   
        for (edge = LB; edge < UB; ++edge)
            degrees[iter] += global_values_dev_ptr[valsAndColsOffset + edge];
        // Maybe this can be done asyncronously or after the whole array has been filled
        // I definitely need the sync threads since I modify the shared memory for memcpy
        // alternatively I could create two shared mem arrays, one for async write to global
        // and 1 for reduction
        global_degrees_dev_ptr[degreesOffset + iter] = degrees[iter];
    }
    __syncthreads();
    if (threadIdx.x == 0){
        for (int i = 0; i < numberOfRows; ++i)
            printf("leafIndex %d, blockID %d vertex %d, degree %d\n", leafIndex, blockIdx.x, i, degrees[i]);
        printf("\n");
    }
    int halvedArray = numberOfRows/2;
    while (halvedArray != 0) {
        // Neccessary since numberOfRows is likely greater than blockSize
        for (iter = row; iter < halvedArray; iter += blockDim.x){
            if (iter < halvedArray){
                degrees[iter] += degrees[iter + halvedArray];
            }
        }
        __syncthreads();
        halvedArray /= 2;
    }
    if (row == 0)
        global_edges_left_to_cover_count[leafIndex] = degrees[0];
}

__global__ void InduceRowOfSubgraphs( int numberOfRows,
                                      int levelOffset,
                                      int levelUpperBound,
                                      int numberOfEdgesPerGraph,
                                      int * global_edges_left_to_cover_count,
                                      int * global_row_offsets_dev_ptr,
                                      int * global_columns_dev_ptr,
                                      int * global_values_dev_ptr
                                    ){

    int leafIndex = levelOffset + blockIdx.x;
    if (leafIndex >= levelUpperBound) return;
    int rowOffsOffset;
    if (levelOffset == 0)
        rowOffsOffset = 0;
    else
        rowOffsOffset = (leafIndex-1)/3 * (numberOfRows + 1);
    int valsAndColsOffset = leafIndex * numberOfEdgesPerGraph;

    // Since three children share a parent, it is sensible for the old pointers to be shared memory
    // and for each block to induce three children
    // For now it still global..
    int * old_row_offsets_dev = &(global_row_offsets_dev_ptr[rowOffsOffset]);
    int * old_columns_dev = &(global_columns_dev_ptr[valsAndColsOffset]);
    int * old_values_dev = &(global_values_dev_ptr[valsAndColsOffset]);
    int * new_row_offsets_dev = &(global_row_offsets_dev_ptr[rowOffsOffset]);


    inner_array_t *C_ref = new inner_array_t[numberOfRows];
    for (int child = 1; child <= 3; ++child){
        int row = threadIdx.x;
        int newValsAndColsOffset = (3*leafIndex + child) * numberOfEdgesPerGraph;

        int * new_columns_dev =  &(global_columns_dev_ptr[newValsAndColsOffset]);
        int * new_values_dev = &(global_values_dev_ptr[newValsAndColsOffset]);
        for (int iter = row; iter < numberOfRows; iter += blockDim.x){

            //printf("Thread %d, row %d", threadIdx.x, iter);
            C_ref[iter][0] = 0;
            C_ref[iter][1] = 0;
            //printf("Thread %d, row %d, old_row_offsets_dev[iter] = %d", threadIdx.x, iter, old_row_offsets_dev[iter]);
            //printf("Thread %d, row %d, old_row_offsets_dev[iter+1] = %d", threadIdx.x, iter, old_row_offsets_dev[iter+1]);
            //printf("Thread %d, row %d, old_values_dev[endOffset] = %d", threadIdx.x, iter, old_values_dev[old_row_offsets_dev[iter+1]]);

            int beginIndex = old_row_offsets_dev[iter];
            int endIndex = old_row_offsets_dev[iter+1];

            for (int i = beginIndex; i < endIndex; ++i){
                ++C_ref[iter][old_values_dev[i]];
            }

            // This is  [old degree - new degree , new degree]
            for (int i = 1; i < 2; ++i){
                C_ref[iter][i] = C_ref[iter][i] + C_ref[iter][i-1];
            }
           // printf("Thread %d, row %d, almost done", threadIdx.x, iter);

            /* C_ref[A_row_indices[i]]]-1 , because the values of C_ref are from [1, n] -> [0,n) */
            for (int i = endIndex-1; i >= beginIndex; --i){
                if (old_values_dev[i]){
                    new_columns_dev[new_row_offsets_dev[iter] - C_ref[iter][0] + C_ref[iter][1]-1] = old_columns_dev[i];
                    new_values_dev[new_row_offsets_dev[iter] - C_ref[iter][0] + C_ref[iter][1]-1] = old_values_dev[i];
                    --C_ref[iter][old_values_dev[i]];
                }
            }
            //printf("Thread %d, row %d, finished", threadIdx.x, iter);
        }
        __syncthreads();
        if (threadIdx.x == 0){
            printf("Block %d, levelOffset %d, leafIndex %d, induced child %d\n", blockIdx.x, levelOffset, leafIndex, 3*leafIndex + child);
            for (int i = 0; i < global_edges_left_to_cover_count[leafIndex]; ++i){
                printf("%d ",new_columns_dev[i]);
            }
            printf("\n");
            for (int i = 0; i < global_edges_left_to_cover_count[leafIndex]; ++i){
                printf("%d ",new_values_dev[i]);
            }
            printf("\n");
        }
    }
    delete[] C_ref;
}

__global__ void CalculateNewRowOffsets( int numberOfRows,
                                        int levelOffset,
                                        int levelUpperBound,
                                        int * global_row_offsets_dev_ptr,
                                        int * global_degrees_dev_ptr){
    int threadID = threadIdx.x + blockDim.x * blockIdx.x;
    int leafIndex = levelOffset + threadID;
    if (leafIndex >= levelUpperBound) return;
    int rowOffsOffset = leafIndex * (numberOfRows + 1);
    int degreesOffset = leafIndex * numberOfRows;

    int i = 0;
    printf("leafIndex %d, degreesOffset = %d\n", leafIndex, degreesOffset);
    printf("leafIndex %d, rowOffsOffset = %d\n", leafIndex, rowOffsOffset);
    printf("leafIndex %d, new_row_offsets_dev[%d] = %d\n", leafIndex, i, global_row_offsets_dev_ptr[rowOffsOffset]);

    global_row_offsets_dev_ptr[rowOffsOffset] = i;
    for (i = 1; i <= numberOfRows; ++i)
    {
        global_row_offsets_dev_ptr[rowOffsOffset + i] = global_degrees_dev_ptr[degreesOffset + i - 1] + global_row_offsets_dev_ptr[rowOffsOffset + i - 1];
        printf("leafIndex %d, new_row_offsets_dev[%d] = %d\n", leafIndex, i, global_row_offsets_dev_ptr[rowOffsOffset + i]);
        printf("leafIndex %d, global_row_offsets_dev_ptr[rowOffsOffset + %d - 1] = %d\n", leafIndex, i, global_row_offsets_dev_ptr[rowOffsOffset + i - 1]);
        printf("leafIndex %d, global_degrees_dev_ptr[degreesOffset + %d - 1] = %d\n", leafIndex, i, global_degrees_dev_ptr[degreesOffset + i - 1]);
    }
}

__global__ void First_Graph_GPU(int vertexCount, 
                                int size,
                                int numberOfRows,
                                int * old_row_offsets_dev,
                                int * old_columns_dev,
                                int * old_values_dev,
                                int * new_row_offsets_dev,
                                int * new_columns_dev,
                                int * new_values_dev,
                                int * old_degrees_dev,
                                int * new_degrees_dev,
                                int * global_row_offsets_dev_ptr,
                                int * global_columns_dev_ptr,
                                int * global_values_dev_ptr,
                                int * global_degrees_dev_ptr
                                ) {
/*
        InduceSubgraph(
        numberOfRows,
        old_row_offsets_dev,
        old_columns_dev,
        old_values_dev,
        global_row_offsets_dev_ptr,
        global_columns_dev_ptr); */

     return;
}

// Single thread per leaf
__global__ void CreateSubsetOfRemainingVerticesLevelWise(int levelOffset,
                                                int levelUpperBound,
                                                int numberOfRows,
                                                int * global_degrees_dev_ptr,
                                                int * global_vertices_remaining,
                                                int * global_vertices_remaining_count){
    int threadID = threadIdx.x + blockDim.x * blockIdx.x;
    int leafIndex = levelOffset + threadID;
    if (leafIndex >= levelUpperBound) return;
    int degreesOffset = leafIndex * numberOfRows;

    global_vertices_remaining_count[leafIndex] = 0;

    for (int i = 0; i < numberOfRows; ++i){
        printf("Thread %d, global_degrees_dev_ptr[degreesOffset+%d] : %d\n", threadID, i, global_degrees_dev_ptr[degreesOffset+i]);
        if (global_degrees_dev_ptr[degreesOffset+i] == 0){
            continue;
        } else {
            global_vertices_remaining[degreesOffset+global_vertices_remaining_count[leafIndex]] = i;
            printf("Thread %d, global_vertices_remaining[degreesOffset+%d] : %d\n", threadID, global_vertices_remaining_count[leafIndex], global_vertices_remaining[degreesOffset+global_vertices_remaining_count[leafIndex]]);
            ++global_vertices_remaining_count[leafIndex];
        }
    }
}

// Single thread per leaf
__global__ void DFSLevelWise(int levelOffset,
                            int levelUpperBound,
                            int numberOfRows,
                            int maxDegree,
                            int numberOfEdgesPerGraph,
                            int * global_degrees_dev_ptr,
                            int * global_row_offsets_dev_ptr,
                            int * global_columns_dev_ptr,
                            int * global_values_dev_ptr,
                            int * global_vertices_remaining,
                            int * global_vertices_remaining_count,
                            int * global_paths_ptr,
                            int * global_paths_length,
                            int * global_outgoing_edge_vertices,
                            int * global_outgoing_edge_vertices_count){
    int threadID = threadIdx.x + blockDim.x * blockIdx.x;
    int leafIndex = levelOffset + threadID;
    if (leafIndex >= levelUpperBound) return;
    int degreesOffset = leafIndex * numberOfRows;
    int pathsOffset = leafIndex * 4;
    int rowOffsOffset = leafIndex * (numberOfRows + 1);
    int valsAndColsOffset = leafIndex * numberOfEdgesPerGraph;
    int outgoingEdgeOffset = leafIndex * maxDegree;

    unsigned int counter = 0;
    ulong seed = 0;
    int randomNumber = randomGPU(counter, leafIndex, seed);
    int randomIndex = randomNumber % global_vertices_remaining_count[leafIndex];
    int randomVertex = global_vertices_remaining[degreesOffset+randomIndex];
    global_paths_ptr[pathsOffset + 0] = randomVertex;
    global_paths_length[leafIndex]++;
    printf("Thread %d, randomVertex : %d, path position : %d\n\n", threadID, randomVertex, 0);
// dfs 
    for (int i = 1; i < 4; ++i){
        global_outgoing_edge_vertices_count[leafIndex] = 0;
        for (int j = global_row_offsets_dev_ptr[rowOffsOffset + randomVertex]; 
                j < global_row_offsets_dev_ptr[rowOffsOffset + randomVertex + 1]; ++j){
            printf("Thread %d, global_values_dev_ptr[valsAndColsOffset + %d] : %d\n", threadID, j, global_values_dev_ptr[valsAndColsOffset + j]);
            if (global_values_dev_ptr[valsAndColsOffset + j] == 0)
                continue;
            else {
                global_outgoing_edge_vertices[outgoingEdgeOffset+global_outgoing_edge_vertices_count[leafIndex]] = global_columns_dev_ptr[valsAndColsOffset + j];
                printf("Thread %d, global_outgoing_edge_vertices[outgoingEdgeOffset+%d] : %d\n", threadID, global_outgoing_edge_vertices_count[leafIndex], global_outgoing_edge_vertices[outgoingEdgeOffset+global_outgoing_edge_vertices_count[leafIndex]]);
                ++global_outgoing_edge_vertices_count[leafIndex];
            }
        }
        ++counter;
        randomNumber = randomGPU(counter, leafIndex, seed);
        randomIndex = randomNumber % global_outgoing_edge_vertices_count[leafIndex];
        randomVertex = global_columns_dev_ptr[valsAndColsOffset + global_outgoing_edge_vertices[outgoingEdgeOffset+randomIndex]];
        
        if (randomVertex == global_paths_ptr[pathsOffset + i - 1]){
            // if degree is greater than 1 there exists an alternative path 
            // which doesnt form a simple cycle
            if (global_degrees_dev_ptr[degreesOffset+randomVertex] > 1){
                // Non-deterministic time until suitable edge which 
                // doesn't form a simple cycle is found.
                while(randomVertex == global_paths_ptr[pathsOffset + i - 1]){
                    ++counter;
                    randomNumber = randomGPU(counter, leafIndex, seed);
                    randomIndex = randomNumber % global_outgoing_edge_vertices_count[leafIndex];
                    randomVertex = global_columns_dev_ptr[valsAndColsOffset + global_outgoing_edge_vertices[outgoingEdgeOffset+randomIndex]];
                }
            } else {
                break;
            }
        }
        global_paths_ptr[pathsOffset + i] = randomVertex;
        ++global_paths_length[leafIndex];
        printf("Thread %d, randomVertex : %d, path position : %d\n\n", threadID, randomVertex, i);
        printf("Thread %d, global_paths_ptr[pathsOffset + %d] : %d", threadID, i, global_paths_ptr[pathsOffset + i]);
    }
}

__global__ void GetRandomVertex(int levelOffset,
                                int levelUpperBound,
                                int numberOfRows,
                                int * global_remaining_vertices_dev_ptr,
                                int * global_remaining_vertices_size_dev_ptr,
                                int * global_paths_ptr){

    int threadID = threadIdx.x + blockDim.x * blockIdx.x;
    // Since each thread calculates four random numbers
    int leafIndex = levelOffset + threadID * 4;
    if (leafIndex >= levelUpperBound) return;

    RNG::ctr_type r;
    unsigned int counter = 0;
    ulong seed = 0;
    int remainingVertsOffset, pathsOffset, iteration, remainingVerticesSize; 
    remainingVertsOffset = leafIndex * numberOfRows;
    pathsOffset = leafIndex * 4;
    // r contains 4 random ints
    r = randomGPU_four(counter, leafIndex, seed);
    for(iteration = 0; iteration < 4 && (leafIndex + iteration) < levelUpperBound; ++iteration){
        remainingVerticesSize = global_remaining_vertices_size_dev_ptr[leafIndex];
        global_paths_ptr[pathsOffset] = r[iteration] % remainingVerticesSize;
        printf("Thread %d, leafIndex %d, random vertex %d", threadID, leafIndex, global_paths_ptr[pathsOffset]);
        remainingVertsOffset += numberOfRows;
        pathsOffset += 4;
    }
}

__global__ void GetRandomVertexSharedMem(int levelOffset,
                                int levelUpperBound,
                                int numberOfRows,
                                int * global_remaining_vertices_dev_ptr,
                                int * global_remaining_vertices_size_dev_ptr,
                                int * global_paths_ptr){

    int threadID = threadIdx.x + blockDim.x * blockIdx.x;
    // Since each thread calculates four random numbers
    int leafIndex = levelOffset + threadID * 4;
    extern __shared__ RNG::ctr_type random123Objects[];
    unsigned int counter = 0;
    ulong seed = 0;
    int r123Index = threadIdx.x / 4;
    
    if (threadIdx.x % 4 == 0)
       random123Objects[r123Index] = randomGPU_four(counter, leafIndex, seed);
    __syncthreads();

    if(leafIndex < levelUpperBound){
        int remainingVertsOffset = leafIndex * numberOfRows;
        int pathsOffset = leafIndex * 4;
        int randomNumIndex = threadIdx.x % 4;
        int remainingVerticesSize = global_remaining_vertices_size_dev_ptr[remainingVertsOffset];
        global_paths_ptr[pathsOffset] = (random123Objects[r123Index])[randomNumIndex] % remainingVerticesSize;
    }
}

__global__ void ParallelDFSRandom(int levelOffset,
                            int levelUpperBound,
                            int numberOfRows,
                            int numberOfEdgesPerGraph,
                            int * global_row_offsets_dev_ptr,
                            int * global_columns_dev_ptr,
                            int * global_remaining_vertices_dev_ptr,
                            int * global_remaining_vertices_size_dev_ptr,
                            int * global_degrees_dev_ptr,
                            int * global_paths_ptr,
                            int * global_nonpendant_path_bool_dev_ptr,
                            int * global_nonpendant_child_dev_ptr){
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Entered DFS\n");
        printf("\n");
    }
    int leafIndex = levelOffset + blockIdx.x;
    if (leafIndex >= levelUpperBound)
        return;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Set leafIndex\n");
        printf("\n");
    }
    // Initialized to 0, so will always perform DFS on first call
    // Subsequently, only perform DFS on pendant edges, so nonpendant false
    if (global_nonpendant_path_bool_dev_ptr[leafIndex])
        return;
    int globalPathOffset = leafIndex * 4;
    int sharedMemPathOffset = threadIdx.x * 4;
    int rowOffsOffset = leafIndex * (numberOfRows + 1);
    int valsAndColsOffset = leafIndex * numberOfEdgesPerGraph;
    int degreesOffset = leafIndex * numberOfRows;
    extern __shared__ int pathsAndPendantStatus[];
    int isInvalidPathBooleanArrayOffset = blockDim.x * 4;
    int iteration = 0;
    RNG::ctr_type r;
    unsigned int counter = 0;
    ulong seed = threadIdx.x;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Setup offsets\n");
        printf("\n");
    }
    int remainingVerticesSize = global_remaining_vertices_size_dev_ptr[leafIndex];
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("remainingVerticesSize\n");
        printf("\n");
    }
    int outEdgesCount;
    r = randomGPU_four(counter, leafIndex, seed);
    // Random starting point
    pathsAndPendantStatus[sharedMemPathOffset + iteration] = r[iteration] % remainingVerticesSize;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("pathsAndPendantStatus\n");
        printf("\n");
    }
    ++iteration;

    // Set random out at depth 1
    int randomVertRowOff = global_row_offsets_dev_ptr[rowOffsOffset + pathsAndPendantStatus[sharedMemPathOffset + iteration - 1]];
    // Using degrees allow us to ignore the edges which have been turned off
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("randomVertRowOff %d\n", randomVertRowOff);
        printf("\n");
    }
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("degreesOffset + pathsAndPendantStatus[sharedMemPathOffset + iteration - 1] %d\n",degreesOffset + pathsAndPendantStatus[sharedMemPathOffset + iteration - 1]);
    }
       if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("degreesOffset  %d\n",degreesOffset);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("pathsAndPendantStatus[sharedMemPathOffset + iteration - 1] %d\n",pathsAndPendantStatus[sharedMemPathOffset + iteration - 1]);
    }
    outEdgesCount = global_degrees_dev_ptr[degreesOffset + pathsAndPendantStatus[sharedMemPathOffset + iteration - 1]];
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("outEdgesCount %d\n", outEdgesCount);
        printf("\n");
    }
    //outEdgesCount = global_row_offsets_dev_ptr[rowOffsOffset + pathsAndPendantStatus[sharedMemPathOffset + iteration - 1] + 1]
    //                - randomVertRowOff;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("valsAndColsOffset + randomVertRowOff + (r[iteration] mod outEdgesCount) %d\n", valsAndColsOffset + randomVertRowOff + (r[iteration] % outEdgesCount));
    }
    pathsAndPendantStatus[sharedMemPathOffset + iteration] =  global_columns_dev_ptr[valsAndColsOffset + randomVertRowOff + (r[iteration] % outEdgesCount)];
    ++iteration;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Block %d, levelOffset %d, leafIndex %d, got through first 2 iterations\n", blockIdx.x, levelOffset, leafIndex);
        printf("\n");
    }
    // Depth 2 and 3
    for (; iteration < 4; ++iteration){
        randomVertRowOff = global_row_offsets_dev_ptr[rowOffsOffset + pathsAndPendantStatus[sharedMemPathOffset + iteration - 1]];
        // Using degrees allow us to ignore the edges which have been turned off
        outEdgesCount = global_degrees_dev_ptr[degreesOffset + pathsAndPendantStatus[sharedMemPathOffset + iteration - 1]];
        //outEdgesCount = global_row_offsets_dev_ptr[rowOffsOffset + pathsAndPendantStatus[sharedMemPathOffset + iteration - 1] + 1]
        //                - randomVertRowOff;
        pathsAndPendantStatus[sharedMemPathOffset + iteration] =  global_columns_dev_ptr[valsAndColsOffset + randomVertRowOff + (r[iteration] % outEdgesCount)];
        // OutEdgesCount != 2 means there is another path that isn't a cycle
        if(pathsAndPendantStatus[sharedMemPathOffset + iteration] == 
            pathsAndPendantStatus[sharedMemPathOffset + iteration - 2]
                && outEdgesCount != 1){
            pathsAndPendantStatus[sharedMemPathOffset + iteration] =  global_columns_dev_ptr[valsAndColsOffset + randomVertRowOff + ((r[iteration] + 1) % outEdgesCount)];
        }
    }
    pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + threadIdx.x] = (pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + 0] == pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + 2]);
    pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + threadIdx.x] |= (pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + 1] == pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + 3]);
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Block %d, levelOffset %d, leafIndex %d, got through last 2 iterations\n", blockIdx.x, levelOffset, leafIndex);
        printf("\n");
    }
    printf("Thread %d is %s\n", threadIdx.x, pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + threadIdx.x] ? "pendant" : "nonpendant");

    int i = blockDim.x/2;
    // Checks for any nonpendant edge path exists
    while (i != 0) {
        if (threadIdx.x < i){
            pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + threadIdx.x] &= pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    // Write pendant status to global memory
    // We detected pendant, but we store the converse, nonpendantness
    if (threadIdx.x == 0){
        global_nonpendant_path_bool_dev_ptr[leafIndex] = !pathsAndPendantStatus[isInvalidPathBooleanArrayOffset];
        printf("leafIndex %d is %s\n", leafIndex, global_nonpendant_path_bool_dev_ptr[leafIndex] ? "nonpendant" : "pendant");
    }
    // A nonpendant exists
    if (!pathsAndPendantStatus[isInvalidPathBooleanArrayOffset]){
        // Regenerate pendant booleans
        pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + threadIdx.x] = (pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + 0] == pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + 2]);
        pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + threadIdx.x] |= (pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + 1] == pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + 3]);
        // We give Case 3 priority over Case 2,
        // Since the serial algorithm short-circuits 
        // upon finding a pendant edge

        // We know either 
        // Case 3 - length 2
        // v, v1
        //path[0] == path[2], desired child is v
        // If path[0] == path[2] then path[0] != path[2]
        // Hence, cI == 0, since false casted to int is 0
        // Therefore, v == path[cI]
        int childIndex = global_paths_ptr[globalPathOffset + 0] != global_paths_ptr[globalPathOffset + 2];

        // or
        // Case 2 - length 3
        // v, v1, v2
        // if path[0] != path[2] was true, then path[1] == path[3]
        // cI == 1, since true casted to int is 1
        // Desired child is v1
        // Therefore, v1 == path[cI]
        global_nonpendant_child_dev_ptr[blockIdx.x] = global_paths_ptr[globalPathOffset + childIndex];
        if (threadIdx.x == 0){
            // One of these is guarunteed nonpendant
            for (int i = 0; i < blockDim.x; ++i){
                // Since these are random paths, using the first possible nonpendant isnt biased
                if (!pathsAndPendantStatus[isInvalidPathBooleanArrayOffset + i]){
                    for (int j = 0; j < 4; ++j)
                        global_paths_ptr[globalPathOffset + j] = pathsAndPendantStatus[i*4 + j];
                    return;
                }
            }
        }
    } else {
        if (threadIdx.x == 0){
            // Since these are random paths, using the first pendant isnt biased
            for (int j = 0; j < 4; ++j){
                global_paths_ptr[globalPathOffset + j] = pathsAndPendantStatus[j];
            }
            printf("leaf index %d, path (%d -> %d -> %d -> %d)\n", leafIndex, 
                global_paths_ptr[globalPathOffset + 0],
                global_paths_ptr[globalPathOffset + 1],
                global_paths_ptr[globalPathOffset + 2],
                global_paths_ptr[globalPathOffset + 3]);
        }
    }
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Block ID %d Finished DFS\n", blockIdx.x);
        printf("\n");
    }
}

__global__ void ParallelProcessPendantEdges(int levelOffset,
                            int levelUpperBound,
                            int numberOfRows,
                            int numberOfEdgesPerGraph,
                            int * global_row_offsets_dev_ptr,
                            int * global_columns_dev_ptr,
                            int * global_values_dev_ptr,
                            int * global_remaining_vertices_dev_ptr,
                            int * global_remaining_vertices_size_dev_ptr,
                            int * global_degrees_dev_ptr,
                            int * global_paths_ptr,
                            int * global_nonpendant_path_bool_dev_ptr){

    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Block ID %d Started ParallelProcessPendantEdges\n", blockIdx.x);
        printf("\n");
    }
    int leafIndex = levelOffset + blockIdx.x;
    // Only process pendant edges
    if (global_nonpendant_path_bool_dev_ptr[leafIndex])
        return;
    int pathsOffset = leafIndex * 4;
    int rowOffsOffset = leafIndex * (numberOfRows + 1);
    int valsAndColsOffset = leafIndex * numberOfEdgesPerGraph;
    int degreesOffset = leafIndex * numberOfRows;
    int childIndex = global_paths_ptr[pathsOffset + 0] != global_paths_ptr[pathsOffset + 2];
    int child = global_paths_ptr[pathsOffset + childIndex];
    int LB, UB, v, vLB, vUB;
    // Set out-edges
    LB = global_row_offsets_dev_ptr[rowOffsOffset + child];
    UB = global_row_offsets_dev_ptr[rowOffsOffset + child + 1]; 
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("block ID %d Set offsets in PPP\n", blockIdx.x);
        printf("\n");
    }   
    for (int edge = LB + threadIdx.x; edge < UB; edge += blockDim.x){
        // Since there are only 2 edges b/w each node,
        // We can safely decrement the target node's degree
        global_degrees_dev_ptr[degreesOffset + 
            global_columns_dev_ptr[valsAndColsOffset + edge]] 
                -= global_values_dev_ptr[valsAndColsOffset + edge];
        global_values_dev_ptr[valsAndColsOffset + edge] = 0;
    }

    if (threadIdx.x == 0){
            global_degrees_dev_ptr[degreesOffset + child] = 0;
    }
    __syncthreads();
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Block ID %d Finished out edges PPP\n", blockIdx.x);
        printf("\n");
    }  
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Block %d, levelOffset %d, leafIndex %d, child removed %d\n", blockIdx.x, levelOffset, leafIndex, child);
        printf("\n");
    }
    // (u,v) is the form of edge pairs.  We are traversing over v's outgoing edges, 
    // looking for u as the destination and turning off that edge.
    // this may be more elegantly handled by 
    // (1) an associative data structure
    // (2) an undirected graph 
    // Parallel implementations of both of these need to be investigated.
    bool foundChild, tmp;
    LB = global_row_offsets_dev_ptr[rowOffsOffset + child];
    UB = global_row_offsets_dev_ptr[rowOffsOffset + child + 1];    // Set out-edges
    for (int edge = LB + threadIdx.x; edge < UB; edge += blockDim.x){
        v = global_columns_dev_ptr[valsAndColsOffset + edge];
        // guarunteed to only have one incoming and one outgoing edge connecting (x,y)
        // All outgoing edges were set and are separated from this method by a __syncthreads
        // Thus there is no chance of decrementing the degree of the same node simulataneously
        vLB = global_row_offsets_dev_ptr[rowOffsOffset + v];
        vUB = global_row_offsets_dev_ptr[rowOffsOffset + v + 1];
        for (int outgoingEdgeOfV = vLB + threadIdx.x; 
                outgoingEdgeOfV < vUB; 
                    outgoingEdgeOfV += blockDim.x){

                foundChild = child == global_columns_dev_ptr[valsAndColsOffset + outgoingEdgeOfV];
                // Set in-edge
                // store edge status
                tmp = global_values_dev_ptr[valsAndColsOffset + outgoingEdgeOfV];
                //   foundChild     tmp   (foundChild & tmp)  (foundChild & tmp)^tmp
                //1)      0          0            0                       0
                //2)      1          0            0                       0
                //3)      0          1            0                       1
                //4)      1          1            1                       0
                //
                // Case 1: isnt child and edge is off, stay off
                // Case 2: is child and edge is off, stay off
                // Case 3: isn't child and edge is on, stay on
                // Case 4: is child and edge is on, turn off
                // All this logic is necessary because we aren't using degree to set upperbound
                // we are using row offsets, which may include some edges turned off on a previous
                // pendant edge processing step.
                global_values_dev_ptr[valsAndColsOffset + outgoingEdgeOfV] ^= (foundChild & tmp);
        
        }
    }
    __syncthreads();
}

__global__ void ParallelProcessDegreeZeroVertices(int levelOffset,
                            int levelUpperBound,
                            int numberOfRows,
                            int * global_remaining_vertices_dev_ptr,
                            int * global_remaining_vertices_size_dev_ptr,
                            int * global_degrees_dev_ptr){

    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Entered ProcessDeg0\n");
        printf("\n");
    }
    int leafIndex = levelOffset + blockIdx.x;
    if (leafIndex >= levelUpperBound)
        return;    

    extern __shared__ int degreeZeroVertex[];

    int degreesOffset = leafIndex * numberOfRows;
    int vertexOffset = 0;
    int numVertices = global_remaining_vertices_size_dev_ptr[leafIndex];
    int numVerticesRemoved = 0;
    for (int vertex = vertexOffset + threadIdx.x; vertex < numVertices; vertexOffset += blockDim.x){
        // Reinitialize
        degreeZeroVertex[threadIdx.x] = 0;
        degreeZeroVertex[threadIdx.x] = (0 == global_degrees_dev_ptr[degreesOffset + global_remaining_vertices_dev_ptr[degreesOffset + vertex]]);
        // Makes this entry INT_MAX if degree 0
        // Leaves unaltered if not degree 0
        global_remaining_vertices_dev_ptr[degreesOffset + vertex] += (INT_MAX - global_remaining_vertices_dev_ptr[degreesOffset + vertex])*degreeZeroVertex[threadIdx.x];
        int i = blockDim.x/2;
        __syncthreads();
        // Checks for any nonpendant edge path exists
        while (i != 0) {
            if (threadIdx.x < i){
                degreeZeroVertex[threadIdx.x] += degreeZeroVertex[threadIdx.x + i];
            }
            __syncthreads();
            i /= 2;
        }
        if (threadIdx.x == 0)
            numVerticesRemoved += degreeZeroVertex[threadIdx.x];
    }
    // Update remaining vert size
    // Now just need to sort those INT_MAX entries to the end of the array
    if (threadIdx.x == 0)
        global_remaining_vertices_size_dev_ptr[leafIndex] -= numVerticesRemoved;
}


__global__ void ParallelCreateLevelAwareRowOffsets(int levelOffset,
                            int levelUpperBound,
                            int numberOfRows,
                            int numberOfEdgesPerGraph,
                            int * global_row_offsets_dev_ptr,
                            int * global_offsets_buffer){

    int leafIndex = levelOffset + blockIdx.x;
    if (leafIndex >= levelUpperBound)
        return;    

    int rowOffsOffset = leafIndex * (numberOfRows + 1);
    int bufferRowOffsOffset = blockIdx.x * (numberOfRows + 1);

    for (int iter = threadIdx.x; iter < numberOfRows+1; iter += blockDim.x){
        global_offsets_buffer[bufferRowOffsOffset + iter] = (blockIdx.x * numberOfEdgesPerGraph) + global_row_offsets_dev_ptr[rowOffsOffset + iter];
    }
}

__global__ void ParallelQuicksortWithDNF(int levelOffset,
                            int levelUpperBound,
                            int numberOfRows,
                            int numberOfEdgesPerGraph,
                            int * global_row_offsets_dev_ptr,
                            int * global_columns_dev_ptr,
                            int * global_values_dev_ptr,
                            int * global_degrees_dev_ptr){

    int row = threadIdx.x;



    for (int iter = row; iter < numberOfRows; iter += blockDim.x){

    }

}

/*
__global__ void SerialProcessPendantEdge(int levelOffset,
                            int levelUpperBound,
                            int numberOfRows,
                            int numberOfEdgesPerGraph,
                            int * global_row_offsets_dev_ptr,
                            int * global_columns_dev_ptr,
                            int * global_remaining_vertices_dev_ptr,
                            int * global_remaining_vertices_size_dev_ptr,
                            int * global_paths_ptr,
                            int * global_nonpendant_path_bool_dev_ptr){
    // Set out-edges
    for (int i = 0; i < 2; ++i){
        LB = global_row_offsets_dev_ptr[rowOffsOffset + children[i]];
        UB = global_row_offsets_dev_ptr[rowOffsOffset + children[i] + 1];    
        for (int edge = LB + threadIndex; edge < UB; edge += blockDim.x){
            global_values_dev_ptr[valsAndColsOffset + edge] = 0;
        }
    }
    __syncthreads();
    if (threadIndex == 0 && blockIdx.x == 0){
        printf("Block %d, levelOffset %d, leafIndex %d, children removed %d %d\n", blockIdx.x, levelOffset, leafIndex, children[0], children[1]);
        for (int i = 0; i < global_edges_left_to_cover_count[(leafIndex-1)/3]; ++i){
            printf("(%d, %d) ",global_columns_dev_ptr[valsAndColsOffset + i], global_values_dev_ptr[valsAndColsOffset + i]);
        }
        printf("\n");
    }
    // (u,v) is the form of edge pairs.  We are traversing over v's outgoing edges, 
    // looking for u as the destination and turning off that edge.
    // this may be more elegantly handled by 
    // (1) an associative data structure
    // (2) an undirected graph 
    // Parallel implementations of both of these need to be investigated.
    for (int i = 0; i < 2; ++i){
        LB = global_row_offsets_dev_ptr[rowOffsOffset + children[i]];
        UB = global_row_offsets_dev_ptr[rowOffsOffset + children[i] + 1];    // Set out-edges
        for (int edge = LB + threadIndex; edge < UB; edge += blockDim.x){
            v = global_columns_dev_ptr[valsAndColsOffset + edge];
            // guarunteed to only have one incoming and one outgoing edge connecting (x,y)
            vLB = global_row_offsets_dev_ptr[rowOffsOffset + v];
            vUB = global_row_offsets_dev_ptr[rowOffsOffset + v + 1];
            for (int outgoingEdgeOfV = vLB + threadIndex; 
                    outgoingEdgeOfV < vUB; 
                        outgoingEdgeOfV += blockDim.x){
                if (children[i] == global_columns_dev_ptr[valsAndColsOffset + outgoingEdgeOfV]){
                    // Set in-edge
                    global_values_dev_ptr[valsAndColsOffset + outgoingEdgeOfV] = 0;
                }
            }
        }
    }
    __syncthreads();
}
*/
__device__ void SetOutgoingEdges(int rowOffsOffset,
                                int valsAndColsOffset,
                                int degreesOffset,
                                int u,
                                int * global_row_offsets_dev_ptr,
                                int * global_columns_dev_ptr,
                                int * global_values_dev_ptr,
                                int * global_degrees_dev_ptr){
    //int rowOffsOffset = leafIndex * (numberOfRows + 1);
    //int valsAndColsOffset = leafIndex * numberOfEdgesPerGraph;
    int uLB = global_row_offsets_dev_ptr[rowOffsOffset + u];
    int uUB = global_row_offsets_dev_ptr[rowOffsOffset + u + 1];    // Set out-edges
    for (int i = uLB; i < uUB; ++i){
        global_values_dev_ptr[valsAndColsOffset + i] = 0;
    }
    global_degrees_dev_ptr[degreesOffset + u] = 0;
}


__device__ void SetIncomingEdges(int rowOffsOffset,
                                int valsAndColsOffset,
                                int degreesOffset,
                                int u,
                                int * global_row_offsets_dev_ptr,
                                int * global_columns_dev_ptr,
                                int * global_values_dev_ptr,
                                int * global_degrees_dev_ptr){
    int v;
    int uLB = global_row_offsets_dev_ptr[rowOffsOffset + u];
    int uUB = global_row_offsets_dev_ptr[rowOffsOffset + u + 1];
    int vLB;
    int vUB;
        // Set out-edges
    for (int i = uLB; i < uUB; ++i){
        v = global_columns_dev_ptr[valsAndColsOffset + i];
        vLB = global_row_offsets_dev_ptr[rowOffsOffset + v];
        vUB = global_row_offsets_dev_ptr[rowOffsOffset + v + 1];
        for (int j = vLB; i < vUB; ++j){
            if(u == global_columns_dev_ptr[valsAndColsOffset + j]){
                global_values_dev_ptr[valsAndColsOffset + j] = 0;
                --global_degrees_dev_ptr[degreesOffset + v];
                break;
            }
        }
    }
}

// Single threaded version
// DFS is implicitly single threaded
__global__ void GenerateChildren(int leafIndex,
                                int numberOfRows,
                                int maxDegree,
                                int numberOfEdgesPerGraph,
                                int * global_row_offsets_dev_ptr,
                                int * global_columns_dev_ptr,
                                int * global_values_dev_ptr,
                                int * global_degrees_dev_ptr,
                                int * global_vertices_remaining,
                                int * global_paths_ptr,
                                int * global_vertices_remaining_count,
                                int * global_outgoing_edge_vertices,
                                int * global_outgoing_edge_vertices_count){
                             
    int threadID = threadIdx.x + blockDim.x * blockIdx.x;
    if(threadID > 0) return;
    printf("Thread %d starting", threadID);

    int pathsOffset = leafIndex * 4;
    int rowOffsOffset = leafIndex * (numberOfRows + 1);
    int valsAndColsOffset = leafIndex * numberOfEdgesPerGraph;
    int degreesOffset = leafIndex * numberOfRows;
    int outgoingEdgeOffset = leafIndex * maxDegree;

    printf("Thread %d, pathsOffset : %d", threadID, pathsOffset);
    printf("Thread %d, rowOffsOffset : %d", threadID, rowOffsOffset);
    printf("Thread %d, valsAndColsOffset %d: ", threadID, valsAndColsOffset);
    printf("Thread %d, degreesOffset : %d", threadID, degreesOffset);

// Get random vertex

    unsigned int counter = 0;
    ulong seed = 0;
    int randomNumber = randomGPU(counter, leafIndex, seed);
    int randomIndex = randomNumber % global_vertices_remaining_count[leafIndex];
    int randomVertex = global_vertices_remaining[degreesOffset+randomIndex];
    printf("Thread %d, randomVertex : %d", threadID, randomVertex);
// dfs 
    for (int i = 0; i < 4; ++i){
        global_paths_ptr[pathsOffset + i] = randomVertex;
        printf("Thread %d, global_paths_ptr[pathsOffset + %d] : %d", threadID, i, global_paths_ptr[pathsOffset + i]);

        if (randomVertex == -1)
            break;
        global_outgoing_edge_vertices_count[leafIndex] = 0;
        for (int j = global_row_offsets_dev_ptr[rowOffsOffset + randomVertex]; 
                j < global_row_offsets_dev_ptr[rowOffsOffset + randomVertex + 1]; ++j){
            printf("Thread %d, global_values_dev_ptr[valsAndColsOffset + %d] : %d\n", threadID, j, global_values_dev_ptr[valsAndColsOffset + j]);
            if (global_values_dev_ptr[valsAndColsOffset + j] == 0)
                continue;
            else {
                global_outgoing_edge_vertices[outgoingEdgeOffset+global_outgoing_edge_vertices_count[leafIndex]] = j;
                printf("Thread %d, global_outgoing_edge_vertices[outgoingEdgeOffset+%d] : %d\n", threadID, global_outgoing_edge_vertices_count[leafIndex], global_outgoing_edge_vertices[outgoingEdgeOffset+global_outgoing_edge_vertices_count[leafIndex]]);
                ++global_outgoing_edge_vertices_count[leafIndex];
            }
        }
        ++counter;
        randomNumber = randomGPU(counter, leafIndex, seed);
        randomIndex = randomNumber % global_outgoing_edge_vertices_count[leafIndex];
        randomVertex = global_outgoing_edge_vertices[outgoingEdgeOffset+randomIndex];
        
        if (i > 0 && randomVertex == global_paths_ptr[pathsOffset + i - 1]){
            if (global_degrees_dev_ptr[degreesOffset+randomVertex] > 1){
                while(randomVertex == global_paths_ptr[pathsOffset + i - 1]){
                    ++counter;
                    randomNumber = randomGPU(counter, leafIndex, seed);
                    randomIndex = randomNumber % global_outgoing_edge_vertices_count[leafIndex];
                    randomVertex = global_outgoing_edge_vertices[outgoingEdgeOffset+randomIndex];
                }
            } else {
                randomVertex = -1;
            }
        }
    }
}

// Fill a perfect 3-ary tree to a given depth
__global__ void PopulateTreeParallelLevelWise_GPU(int numberOfLevels, 
                                                long long edgesPerNode,
                                                long long numberOfVertices,
                                                int * new_row_offsets_dev,
                                                int * new_columns_dev,
                                                int * values_dev,
                                                int * new_degrees_dev){

    long long myLevel = blockIdx.x;

    if (myLevel >= numberOfLevels)
        return;

    long long myLevelSize;
    long long levelOffset;
    if (myLevel != 0){
        myLevelSize = pow(3.0, myLevel-1);
        levelOffset = CalculateLevelOffset(myLevel);
    } else {
        myLevelSize = 1;
        levelOffset = 0;
    }

    long long leafIndex = threadIdx.x;

    for (int node = leafIndex; node < myLevelSize; node += blockDim.x){
        //graphs[levelOffset + node] = new Graph_GPU(g);
        printf("Thread %lu, block %lu", leafIndex, myLevel);

    }
}



void CallPopulateTree(int numberOfLevels, 
                    Graph & g){


    int maxDegree = g.GetLargestDegree();

    //int treeSize = 200000;
    int counters = 2;
    numberOfLevels = 2;
    int deepestLevelSize = CalculateDeepestLevelWidth(numberOfLevels-1);;
    long long treeSize = CalculateSpaceForDesiredNumberOfLevels(numberOfLevels);
    long long bufferSize = deepestLevelSize;
    int expandedData = g.GetEdgesLeftToCover();
    int condensedData = g.GetVertexCount();
    int condensedData_plus1 = condensedData + 1;
    long long sizeOfSingleGraph = expandedData*2 + 2*condensedData + condensedData_plus1 + maxDegree + counters;
    long long totalMem = sizeOfSingleGraph * treeSize * sizeof(int) + 
        condensedData * bufferSize * sizeof(int) +
            2 * expandedData * bufferSize * sizeof(int);

    std::vector<std::vector<int>> pendantChildren(treeSize);
    int pendantChild;

    int num_gpus;
    size_t free, total;
    cudaGetDeviceCount( &num_gpus );
    for ( int gpu_id = 0; gpu_id < num_gpus; gpu_id++ ) {
        cudaSetDevice( gpu_id );
        int id;
        cudaGetDevice( &id );
        cudaMemGetInfo( &free, &total );
        std::cout << "GPU " << id << " memory: free=" << free << ", total=" << total << std::endl;
    }

    std::cout << "You are about to allocate " << double(totalMem)/1024/1024/1024 << " GB" << std::endl;
    std::cout << "Your GPU RAM has " << double(free)/1024/1024/1024 << " GB available" << std::endl;
    do 
    {
        std::cout << '\n' << "Press enter to continue...; ctrl-c to terminate";
    } while (std::cin.get() != '\n');

    int * global_row_offsets_dev_ptr;
    int * global_columns_dev_ptr;
    int * global_values_dev_ptr;
    int * global_degrees_dev_ptr; 
    int * global_paths_ptr; 
    int * global_remaining_vertices_ptr;
    int * global_remaining_vertices_size_dev_ptr;
    int * global_nonpendant_path_bool_dev_ptr;
    int * global_nonpendant_child_dev_ptr;
    int * global_paths_length;
    int * global_edges_left_to_cover_count;

    int * global_column_buffer;
    int * global_vertex_buffer;
    int * global_value_buffer;
    int * global_offsets_buffer;

    int max_dfs_depth = 4;
    int numberOfRows = g.GetNumberOfRows();
    int numberOfEdgesPerGraph = g.GetEdgesLeftToCover(); 
    int verticesRemainingInGraph = g.GetRemainingVertices().size(); 

    cudaMalloc( (void**)&global_row_offsets_dev_ptr, ((numberOfRows+1)*treeSize) * sizeof(int) );
    cudaMalloc( (void**)&global_columns_dev_ptr, (numberOfEdgesPerGraph*treeSize) * sizeof(int) );
    cudaMalloc( (void**)&global_values_dev_ptr, (numberOfEdgesPerGraph*treeSize) * sizeof(int) );
    cudaMalloc( (void**)&global_degrees_dev_ptr, (numberOfRows*treeSize) * sizeof(int) );
    cudaMalloc( (void**)&global_paths_ptr, (max_dfs_depth*treeSize) * sizeof(int) );
    cudaMalloc( (void**)&global_remaining_vertices_ptr, (numberOfRows*treeSize) * sizeof(int) );

    cudaMalloc( (void**)&global_column_buffer, numberOfEdgesPerGraph * deepestLevelSize * sizeof(int) );
    cudaMalloc( (void**)&global_value_buffer, numberOfEdgesPerGraph * deepestLevelSize * sizeof(int) );
    cudaMalloc( (void**)&global_vertex_buffer, numberOfRows * deepestLevelSize * sizeof(int) );
    cudaMalloc( (void**)&global_offsets_buffer, (numberOfRows+1) * deepestLevelSize * sizeof(int) );


    cudaMalloc( (void**)&global_paths_length, treeSize * sizeof(int) );
    cudaMalloc( (void**)&global_remaining_vertices_size_dev_ptr, treeSize * sizeof(int) );
    cudaMalloc( (void**)&global_nonpendant_path_bool_dev_ptr, deepestLevelSize * sizeof(int) );
    cudaMalloc( (void**)&global_nonpendant_child_dev_ptr, deepestLevelSize * sizeof(int) );

    cudaMalloc( (void**)&global_edges_left_to_cover_count, treeSize * sizeof(int) );

    cudaDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);

    CopyGraphToDevice(g,
                    global_row_offsets_dev_ptr,
                    global_columns_dev_ptr,
                    global_values_dev_ptr,
                    global_degrees_dev_ptr,
                    numberOfEdgesPerGraph,
                    global_edges_left_to_cover_count,
                    global_remaining_vertices_ptr,
                    global_remaining_vertices_size_dev_ptr,
                    verticesRemainingInGraph);

    long long levelOffset = 0;
    long long levelUpperBound;
    int numberOfBlocksForOneThreadPerLeaf;
    numberOfLevels = 1;
    bool pendantNodeExists = true;

    int * pendantBools = new int[deepestLevelSize];
    int * pendantChildrenOfLevel = new int[deepestLevelSize];

    // Determine temporary device storage requirements
    int     *global_columns_tree = NULL;
    // Determine temporary device storage requirements
    int     *global_values_tree = NULL;

    for (int level = 0; level < numberOfLevels; ++level){
        levelUpperBound = CalculateLevelUpperBound(level);
        numberOfBlocksForOneThreadPerLeaf = ((levelUpperBound - levelOffset) + threadsPerBlock - 1) / threadsPerBlock;
        // 1 thread per leaf
        std::cout << "Calling DFS - level " << level << std::endl;
        // 1 block per leaf; tries tPB random paths in G
        // Hence threadsPerBlock*4,
        // Each thread checks it's path's pendant status
        // These booleans are reduced in shared memory
        // Hence + threadsPerBlock
        while(pendantNodeExists){
            std::cout << "pendantNodeExists - true " << std::endl;

            // Assumes all edges are turned on.  We need to compress a graph
            // after processing the edges of pendant paths
            int sharedMemorySize = threadsPerBlock*4 + threadsPerBlock;
            ParallelDFSRandom<<<levelUpperBound-levelOffset,threadsPerBlock,sharedMemorySize*sizeof(int)>>>
                                (levelOffset,
                                levelUpperBound,
                                numberOfRows,
                                numberOfEdgesPerGraph,
                                global_row_offsets_dev_ptr,
                                global_columns_dev_ptr,
                                global_remaining_vertices_ptr,
                                global_remaining_vertices_size_dev_ptr,
                                global_degrees_dev_ptr,
                                global_paths_ptr,
                                global_nonpendant_path_bool_dev_ptr,
                                global_nonpendant_child_dev_ptr);
            
            cudaDeviceSynchronize();
            checkLastErrorCUDA(__FILE__, __LINE__);

            pendantNodeExists = false;
            cudaMemcpy(pendantBools, global_nonpendant_path_bool_dev_ptr, deepestLevelSize*sizeof(int), cudaMemcpyDeviceToHost);
            cudaMemcpy(pendantChildrenOfLevel, global_nonpendant_child_dev_ptr, deepestLevelSize*sizeof(int), cudaMemcpyDeviceToHost);
            

            for (int node = levelOffset; node < levelUpperBound; ++node){
                // global_nonpendant_path_bool_dev_ptr was defined as an OR of 
                // 0) path[0] == path[2]
                // 1) path[1] == path[3]
                std::cout << "node " << node << std::endl;
                std::cout << "global_nonpendant_path_bool_dev_ptr[node] " << pendantBools[node] << std::endl;

                std::cout << "!global_nonpendant_path_bool_dev_ptr[node] " << !pendantBools[node] << std::endl;

                if (!pendantBools[node]){
                    std::cout << "node " << node << " is pendant" << std::endl;

                    pendantNodeExists = true;
                    pendantChild = pendantChildrenOfLevel[node];

                    pendantChildren[node].push_back(pendantChild);
                    std::cout << "node " << node << "'s pendantChild " << pendantChild << " was pushed" << std::endl;
                }
            }
            cudaDeviceSynchronize();
            checkLastErrorCUDA(__FILE__, __LINE__);
            // Each node assigned a block,  outgoing and incoming edges of child 
            // from pendant path processed at thread level
            // Block immediately returns if nonpendant path
            ParallelProcessPendantEdges<<<levelUpperBound-levelOffset,threadsPerBlock>>>
                            (levelOffset,
                            levelUpperBound,
                            numberOfRows,
                            numberOfEdgesPerGraph,
                            global_row_offsets_dev_ptr,
                            global_columns_dev_ptr,
                            global_values_dev_ptr,
                            global_remaining_vertices_ptr,
                            global_remaining_vertices_size_dev_ptr,
                            global_degrees_dev_ptr,
                            global_paths_ptr,
                            global_nonpendant_path_bool_dev_ptr);
            cudaDeviceSynchronize();
            checkLastErrorCUDA(__FILE__, __LINE__);
          
            // Create pointer that starts at beginning of level
            // Leaves are indexed from 0; so I need to add the offset
            // of the leaf from the left of the tree * (numberOfRows+1) so the 
            // sorting operation works on an entire level.
            // global_offsets_buffer = &global_row_offsets_dev_ptr[levelOffset*(numberOfRows+1)];
            ParallelCreateLevelAwareRowOffsets<<<levelUpperBound-levelOffset,threadsPerBlock>>>
                                                (levelOffset,
                                                levelUpperBound,
                                                numberOfRows,
                                                numberOfEdgesPerGraph,
                                                global_row_offsets_dev_ptr,
                                                global_offsets_buffer);

            cudaDeviceSynchronize();
            checkLastErrorCUDA(__FILE__, __LINE__);
            
            global_columns_tree = &global_columns_dev_ptr[levelOffset*numberOfEdgesPerGraph];
            global_values_tree = &global_values_dev_ptr[levelOffset*numberOfEdgesPerGraph];

            // Create a set of DoubleBuffers to wrap pairs of device pointers
            cub::DoubleBuffer<int> d_keys(global_columns_tree, global_column_buffer);
            cub::DoubleBuffer<int> d_values(global_values_tree, global_value_buffer);

            // Determine temporary device storage requirements
            void     *d_temp_storage = NULL;
            size_t   temp_storage_bytes = 0;
            int num_items = (levelUpperBound-levelOffset)*numberOfEdgesPerGraph;
            int num_segments = (levelUpperBound-levelOffset)*(numberOfRows+1);

            cub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, d_keys, d_values,
                num_items, num_segments, global_offsets_buffer, global_offsets_buffer + 1);

            cudaDeviceSynchronize();
            checkLastErrorCUDA(__FILE__, __LINE__);

            // Allocate temporary storage
            cudaMalloc(&d_temp_storage, temp_storage_bytes);

            cudaDeviceSynchronize();
            checkLastErrorCUDA(__FILE__, __LINE__);

            // Run sorting operation
            cub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, d_keys, d_values,
                num_items, num_segments, global_offsets_buffer, global_offsets_buffer + 1);

            cudaDeviceSynchronize();
            checkLastErrorCUDA(__FILE__, __LINE__);

            int * printAlt = d_values.Alternate();
            std::cout << "Unsorted" << std::endl;
            for (int i = 0; i < (levelUpperBound-levelOffset)*numberOfEdgesPerGraph; ++i){
                std::cout << printAlt[i] << " ";
            }
            std::cout << std::endl;

            int * printCurr = d_values.Current();

            std::cout << "Sorted" << std::endl;
            for (int i = 0; i < (levelUpperBound-levelOffset)*numberOfEdgesPerGraph; ++i){
                std::cout << printCurr[i] << " ";
            }
            std::cout << std::endl;
        }
        
        cudaDeviceSynchronize();
        checkLastErrorCUDA(__FILE__, __LINE__);
    }

    for (const auto& inner: pendantChildren) { // auto is std::vector<int>
        for (auto e: inner) { // auto is int
            std::cout << e << " ";
        }
        std::cout << std::endl;
    }
    cudaDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);

    cudaFree( global_row_offsets_dev_ptr );
    cudaFree( global_columns_dev_ptr );
    cudaFree( global_values_dev_ptr );
    cudaFree( global_degrees_dev_ptr );
    cudaFree( global_paths_ptr );
    //cudaFree( global_vertices_remaining );
    //cudaFree( global_vertices_remaining_count );
    //cudaFree( global_outgoing_edge_vertices );
    //cudaFree( global_outgoing_edge_vertices_count );
    cudaFree( global_paths_length );
    cudaFree( global_edges_left_to_cover_count );
    cudaDeviceSynchronize();
}

void CopyGraphToDevice( Graph & g,
                        int * global_row_offsets_dev_ptr,
                        int * global_columns_dev_ptr,
                        int * global_values_dev_ptr,
                        int * global_degrees_dev_ptr,
                        int numberOfEdgesPerGraph,
                        int * global_edges_left_to_cover_count,
                        int * global_remaining_vertices_dev_ptr,
                        int * global_remaining_vertices_size_dev_ptr,
                        int verticesRemainingInGraph){

    int * new_degrees_ptr = thrust::raw_pointer_cast(g.GetNewDegRef().data());
    int * vertices_remaining_ptr = thrust::raw_pointer_cast(g.GetRemainingVertices().data());
    // Graph vectors
    cudaMemcpy(global_degrees_dev_ptr, new_degrees_ptr, g.GetNumberOfRows() * sizeof(int),
                cudaMemcpyHostToDevice);
    cudaMemcpy(global_edges_left_to_cover_count, &numberOfEdgesPerGraph, 1 * sizeof(int),
                cudaMemcpyHostToDevice);
    cudaMemcpy(global_remaining_vertices_dev_ptr, vertices_remaining_ptr, g.GetRemainingVertices().size() * sizeof(int),
            cudaMemcpyHostToDevice);         
    cudaMemcpy(global_remaining_vertices_size_dev_ptr, &verticesRemainingInGraph, 1 * sizeof(int),
            cudaMemcpyHostToDevice);    
    cudaDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);
    // CSR vectors
    thrust::device_vector<int> old_row_offsets_dev = *(g.GetCSR().GetOldRowOffRef());
    thrust::device_vector<int> old_column_indices_dev = *(g.GetCSR().GetOldColRef());

    // SparseMatrix vectors
    thrust::device_vector<int> new_values_dev = g.GetCSR().GetNewValRef();
    thrust::device_vector<int> remaining_vertices_dev = g.GetRemainingVertices();
    // CSR pointers
    int * old_row_offsets_dev_ptr = thrust::raw_pointer_cast(old_row_offsets_dev.data());
    int * old_column_indices_dev_ptr = thrust::raw_pointer_cast(old_column_indices_dev.data());
    
    // SparseMatrix pointers
    int * new_values_dev_ptr = thrust::raw_pointer_cast(new_values_dev.data());

    // Currenly only sets the first graph in the cuda memory
    // Might as well be host code
    CalculateNewRowOffsets<<<1,1>>>(g.GetNumberOfRows(),
                                        0,
                                        1,
                                        global_row_offsets_dev_ptr,
                                        global_degrees_dev_ptr); 

    cudaDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);
    // Currenly only sets the first graph in the cuda memory
    InduceSubgraph<<<1,threadsPerBlock>>>(g.GetNumberOfRows(), 
                            g.GetEdgesLeftToCover(),          
                            old_row_offsets_dev_ptr,
                            old_column_indices_dev_ptr,
                            new_values_dev_ptr,
                            global_row_offsets_dev_ptr,
                            global_columns_dev_ptr,
                            global_values_dev_ptr);
    cudaDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);

    thrust::device_ptr<int> back2Host_ptr = thrust::device_pointer_cast(global_columns_dev_ptr);
    thrust::device_vector<int> back2Host(back2Host_ptr, back2Host_ptr + g.GetEdgesLeftToCover());
    
    thrust::host_vector<int> hostFinal = back2Host;
    std::cout << "Priting data copied there and back" << std::endl;
    std::cout << "Size" << g.GetEdgesLeftToCover() << std::endl;
    for (auto & v : hostFinal)
        std::cout << v << " ";
    std::cout << std::endl;
}


// Logic of the tree
    // Every level decreases the number of remaining vertices by at least 2
    // more sophisticate analysis could be performed by analyzing the graph
    // i.e. number of degree 1 vertices, (case 3) - a level decreases by > 2
    // number of degree 2 vertices with a pendant edge (case 2) - a level decreases by > 2
    // number of triangles in a graph (case 1)
    // gPrime is at root of tree
    // This is a 3-ary tree, m = 3
    // if a node has an index i, its c-th child in range {1,…,m} 
    // is found at index m ⋅ i + c, while its parent (if any) is 
    // found at index floor((i-1)/m).

// This method benefits from more compact storage and 
// better locality of reference, particularly during a 
// preorder traversal. The space complexity of this 
// method is O(m^n).  Actually smaller - TODO
// calculate by recursion tree

    // We are setting parent pointers, in case we find space
    // to be a constraint, we are halfway to dynamic trees,
    // we just need to pop a free graph object off a queue 
    // and induce.  
    // We may have no use for iterating over a graph from the root.
/*
__host__ __device__ void PopulateTree(int treeSize, 
                                std::vector<Graph> & graphs,
                                std::vector<int> & answer){
    // ceiling(vertexCount/2) loops
    int result, childVertex;
    for (int i = 0; i < treeSize; ++i){
        result = GenerateChildren(graphs[i]);
        while (graphs[i].GetChildrenVertices().size() == 1){
            graphs[i].ProcessImmediately(graphs[i].GetChildrenVertices().front());
            graphs[i].GetChildrenVertices().clear();
            result = GenerateChildren(graphs[i]);
        }       
        if (result == -1){
            TraverseUpTree(i, graphs, answer);
            return;
        } else {
            for (int c = 1; c <= 3; ++c){
                std::cout << "i : " << i << ", c : " << c << std::endl;
                graphs[3*i + c].InitGPrime(graphs[i], graphs[i].GetChildrenVertices()[c-1]);
            }
        }
    }
}

// Fill a perfect 3-ary tree to a given depth
__host__ __device__ int PopulateTreeParallelLevelWise(int numberOfLevels, 
                                std::vector<Graph> & graphs,
                                std::vector<int> & answer){
    // ceiling(vertexCount/2) loops
    volatile bool flag=false;
    std::vector<long long> resultsFlags;
    long long maximumLevelSize = pow(3.0, numberOfLevels-1);
    resultsFlags.reserve(maximumLevelSize);
    long long leafIndex;
    long long levelOffset = 0;
    long long upperBound = 0;
    long long previousLevelSize = 0;
    long long thisLevelSize = 0;
    long long count = 0;
    for (int level = 0; level < numberOfLevels; ++level){
        // level 0 - [0,1); lvlOff = 0 + 0
        // level 1 - [1,4); lvlOff = 0 + 3^0 = 1
        // level 2 - [4,13);lvlOff = 1 + 3^1 = 4
        if (level != 0){
            previousLevelSize = thisLevelSize;
            levelOffset += previousLevelSize;
        }
        thisLevelSize = pow(3.0, level);
        upperBound = levelOffset + thisLevelSize;
        
        resultsFlags.clear();
        for (count = levelOffset; count < upperBound; ++count)
            resultsFlags.push_back(-1);

//        #pragma omp parallel for default(none) \
//            shared(graphs, levelOffset, level, numberOfLevels, upperBound, flag, resultsFlags) \
//            private (leafIndex)

        for (leafIndex = levelOffset; leafIndex < upperBound; ++leafIndex){
            
            // Allows for pseudo-early termination if an answer is found
            // All iterations which havent begun will terminate quickly
    //        if(flag) continue;

            if (flag) continue;
            int result;
            result = GenerateChildren(graphs[leafIndex]);
            if (result == -1)
            {
                flag = true;
                resultsFlags[leafIndex - levelOffset] = leafIndex;
            }
            // This is a strict 3-ary tree
            while (graphs[leafIndex].GetChildrenVertices().size() == 1){
                graphs[leafIndex].ProcessImmediately(graphs[leafIndex].GetChildrenVertices().front());
                graphs[leafIndex].GetChildrenVertices().clear();
                result = GenerateChildren(graphs[leafIndex]);
                if (result == -1)
                {
                    flag = true;
                    resultsFlags[leafIndex - levelOffset] = leafIndex;
                }  
            }
            // We dont initiate the last level and we stop if we cant make more children 
            if (level + 1 != numberOfLevels && result != -1)
                for (int c = 1; c <= 3; ++c){
                    printf("level : %d, level offset : %lld, leafIndex : %lld, c : %d\n", level, levelOffset, leafIndex, c);
                    graphs[3*leafIndex + c].InitGPrime(graphs[leafIndex], graphs[leafIndex].GetChildrenVertices()[c-1]);
                }
        }
        if (flag)
            for(auto & v : resultsFlags)
                if (v != -1)
                    return v;
    }
    return -1;
}
// This method can be rewritten to use fill all Graphs allocated
// Irrespective of whether the last level is full
__host__ __device__ void PopulateTreeParallelAsymmetric(int treeSize, 
                                std::vector<Graph> & graphs,
                                std::vector<int> & answer){
    // ceiling(vertexCount/2) loops
    int numberOfLevels = int(ceil(log(treeSize) / log(3)));
    int leafIndex;
    int levelOffset = 0;
    int upperBound = 0;
    for (int level = 0; level < numberOfLevels; ++level){
        // level 0 - [0,1); lvlOff = 0 + 0
        // level 1 - [1,4); lvlOff = 0 + 3^0 = 1
        // level 2 - [4,13);lvlOff = 1 + 3^1 = 4
        if (level != 0)
            levelOffset += int(pow(3.0, level-1));
        if (level + 1 != numberOfLevels){
            upperBound = levelOffset + int(pow(3.0, level));
        } else {
            upperBound = treeSize;
        }
        #pragma omp parallel for default(none) \
                            shared(treeSize, graphs, levelOffset, level, upperBound) \
                            private (leafIndex)
        for (leafIndex = levelOffset; leafIndex < upperBound; ++leafIndex){
            int result;
            result = GenerateChildren(graphs[leafIndex]);
            // This is a strict 3-ary tree
            while (graphs[leafIndex].GetChildrenVertices().size() == 1){
                graphs[leafIndex].ProcessImmediately(graphs[leafIndex].GetChildrenVertices().front());
                graphs[leafIndex].GetChildrenVertices().clear();
                result = GenerateChildren(graphs[leafIndex]);
            }       
            for (int c = 1; c <= 3; ++c){
                if (3*leafIndex + c < treeSize){
                    printf("level : %d, level offset : %d, leafIndex : %d, c : %d\n", level, levelOffset, leafIndex, c);
                    graphs[3*leafIndex + c].InitGPrime(graphs[leafIndex], graphs[leafIndex].GetChildrenVertices()[c-1]);
                }
            }
        }
    }
}

__host__ __device__ int GenerateChildren(Graph & child_g){

    std::vector< std::vector<int> > & childrensVertices_ref = child_g.GetChildrenVertices();

    std::vector<int> path;
    int randomVertex = GetRandomVertex(child_g.GetRemainingVerticesRef());
    std::cout << "Grabbing a randomVertex: " <<  randomVertex<< std::endl;
    if(randomVertex == -1)
        return randomVertex;

    path.push_back(randomVertex);

    DFS(child_g.GetCSR().GetNewRowOffRef(), 
        child_g.GetCSR().GetNewColRef(), 
        child_g.GetCSR().GetNewValRef(),
        path, 
        randomVertex);

    for (auto & v : path){
        std::cout << v << " ";
        if (v < 0 || v > child_g.GetVertexCount())
            std::cout << "error" << std::endl;
    }
    std::cout << std::endl;
    int caseNumber = classifyPath(path);
    std::cout << "Case number: " << caseNumber << std::endl;
    createVertexSetsForEachChild(childrensVertices_ref, caseNumber, path);
    for (auto & vv : childrensVertices_ref)
        for (auto & v : vv)
            if (v < 0 || v > child_g.GetVertexCount())
                std::cout << "error" << std::endl;

    return 0;
}

__host__ __device__ int GetRandomVertex(std::vector<int> & verticesRemaining){
    if(verticesRemaining.size() == 0)
        return -1;
    int index = rand() % verticesRemaining.size();
    return verticesRemaining[index];
}


// DFS of maximum length 3. No simple cycles u -> v -> u 
__host__ __device__ void DFS(std::vector<int> & new_row_off,
                    std::vector<int> & new_col_ref, 
                    std::vector<int> & new_vals_ref,
                    std::vector<int> & path, 
                    int rootVertex){
    if (path.size() == 4)
        return;

    int randomOutgoingEdge = GetRandomOutgoingEdge(new_row_off, new_col_ref, new_vals_ref, rootVertex, path);
    if (randomOutgoingEdge < 0) {
        std::cout << "terminate DFS" << std::endl;
        return;
    } else {
        path.push_back(randomOutgoingEdge);
        return DFS(new_row_off, new_col_ref, new_vals_ref, path, randomOutgoingEdge);
    }
}

__host__ __device__ int GetRandomOutgoingEdge(  std::vector<int> & new_row_off,
                                        std::vector<int> & new_col_ref,
                                        std::vector<int> & new_values_ref,
                                        int v, 
                                        std::vector<int> & path){

    std::vector<int> outgoingEdges(&new_col_ref[new_row_off[v]],
                        &new_col_ref[new_row_off[v+1]]);

    std::vector<int> outgoingEdgeValues(&new_values_ref[new_row_off[v]],
                    &new_values_ref[new_row_off[v+1]]);

    std::vector<std::pair<int, int>> edgesAndValues;
    edgesAndValues.reserve(outgoingEdges.size());
    std::transform(outgoingEdges.begin(), outgoingEdges.end(), outgoingEdgeValues.begin(), std::back_inserter(edgesAndValues),
               [](int a, int b) { return std::make_pair(a, b); });

    std::random_device rd;
    std::mt19937 g(rd());
    std::shuffle(edgesAndValues.begin(), edgesAndValues.end(), g);
    std::vector< std::pair<int,int> >::iterator it = edgesAndValues.begin();

    while (it != edgesAndValues.end()){
        // To prevent simple paths, must at least have 2 entries, 
        //assuming there are no self edges, since the first entry, v,
        //is randomly chosen and the second entry is a random out edge 
        if (path.size() > 1 && it->first == path.rbegin()[1]  || it->second == 0) {
            //std::cout << "Wouldve been a simple path, skipping " << *it << std::endl;
            ++it;
        } else
            return it->first;
    }

    return -1;
}


__host__ __device__ int classifyPath(std::vector<int> & path){
    if (path.size()==2)
        return 3;
    else if (path.size()==3)
        return 2;
    else if (path.front() == path.back())
        return 1;
    else
        return 0;
}

__host__ __device__ void createVertexSetsForEachChild(std::vector< std::vector<int> > & childrensVertices,
                                            int caseNumber, 
                                            std::vector<int> & path){
    if (caseNumber == 0) {
        // 3 Children 
        childrensVertices.resize(3);
        // Each with 2 vertices 
        for (auto & cV : childrensVertices)
            cV.reserve(2);
        childrensVertices[0].push_back(path[0]);
        childrensVertices[0].push_back(path[2]);

        childrensVertices[1].push_back(path[1]);
        childrensVertices[1].push_back(path[2]);

        childrensVertices[2].push_back(path[1]);
        childrensVertices[2].push_back(path[3]);

    } else if (caseNumber == 1) {

        // 3 Children 
        childrensVertices.resize(3);
        // Each with 2 vertices 
        for (auto & cV : childrensVertices)
            cV.reserve(2);
        childrensVertices[0].push_back(path[0]);
        childrensVertices[0].push_back(path[1]);

        childrensVertices[1].push_back(path[1]);
        childrensVertices[1].push_back(path[2]);

        childrensVertices[2].push_back(path[0]);
        childrensVertices[2].push_back(path[2]);

    } else if (caseNumber == 2) {

        childrensVertices.resize(1);
        childrensVertices[0].reserve(1);
        childrensVertices[0].push_back(path[1]);

    } else {

        childrensVertices.resize(1);
        childrensVertices[0].reserve(1);
        childrensVertices[0].push_back(path[0]);

    }
}

__host__ __device__ void TraverseUpTree(int index, 
                                std::vector<Graph> & graphs,
                                std::vector<int> & answer){
    bool haventReachedRoot = true;
    while(haventReachedRoot) {
        if (index == 0)
            haventReachedRoot = false;
        for (auto & v : graphs[index].GetVerticesThisGraphIncludedInTheCover())
            answer.push_back(v);
        index = (index-1)/3;
    } 
}
*/

#endif