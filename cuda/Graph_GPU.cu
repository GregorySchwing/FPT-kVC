
#include <hip/hip_runtime.h>
#ifdef FPT_CUDA


#include "Graph_GPU.cuh"


__host__ __device__ Graph_GPU::Graph_GPU(const Graph & other): csr(other.csr),
    vertexCount(other.vertexCount){

    //cudaMalloc(&hasntBeenRemoved, other.vertexCount*sizeof(int)); 
    //cudaMalloc(&verticesRemaining, other.vertexCount*sizeof(int)); 
    //cudaMalloc(&new_degrees_dev, other.vertexCount*sizeof(int)); 
}

__host__ __device__ void Graph_GPU::InitTree(Graph & root,
                                            long long treeSize, 
                                            long long edgesPerNode,
                                            long long numberOfVertices,
                                            int startingLevel, 
                                            int endingLevel, 
                                            Graph ** tree,
                                            int ** new_row_offsets_dev,
                                            int ** new_columns_dev,
                                            int ** values_dev,
                                            int ** new_degrees_dev){

    long long calculatedSizeReq = CalculateSizeRequirement(startingLevel, 
                                                            endingLevel);


    if (treeSize != calculatedSizeReq)
        printf("Asymmetric tree");
    else
        printf("Symmetric tree");


}

__host__ __device__ Graph_GPU::~Graph_GPU(){
    delete new_degrees_dev;
    delete csr.new_row_offsets_dev;
    delete csr.new_column_indices_dev;
    delete csr.new_values_dev;
}

#endif
