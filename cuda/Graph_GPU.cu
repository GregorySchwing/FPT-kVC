#include "Graph_GPU.cuh"

__host__ Graph_GPU::Graph_GPU(const Graph & other): csr(other.csr),
    vertexCount(other.vertexCount){

    hipMalloc(&hasntBeenRemoved, other.vertexCount*sizeof(int)); 
    hipMalloc(&verticesRemaining, other.vertexCount*sizeof(int)); 
    hipMalloc(&new_degrees_dev, other.vertexCount*sizeof(int)); 
}


__host__ __device__ Graph_GPU::~Graph_GPU(){
    csr.~CSR_GPU();
    hipFree(hasntBeenRemoved);
    hipFree(verticesRemaining);
    hipFree(new_degrees_dev);
}