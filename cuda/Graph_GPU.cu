#include "Graph_GPU.cuh"


__host__ __device__ Graph_GPU::Graph_GPU(const Graph & other): csr(other.csr),
    vertexCount(other.vertexCount){

    //hipMalloc(&hasntBeenRemoved, other.vertexCount*sizeof(int)); 
    //hipMalloc(&verticesRemaining, other.vertexCount*sizeof(int)); 
    //hipMalloc(&new_degrees_dev, other.vertexCount*sizeof(int)); 
}


__host__ __device__ void Graph_GPU::InitTree(long long treeSize, 
                                            long long edgesPerNode,
                                            long long numberOfVertices,
                                            int startingLevel, 
                                            int endingLevel, 
                                            Graph ** tree,
                                            int ** new_row_offsets_dev,
                                            int ** new_columns_dev,
                                            int ** values_dev,
                                            int ** new_degrees_dev){

    long long calculatedSizeReq = CalculateSizeRequirement(startingLevel, 
                                                            endingLevel);

    if (treeSize != calculatedSizeReq)
        printf("Asymmetric tree");
    else
        printf("Symmetric tree");



    

    
    //hipMalloc(&hasntBeenRemoved, other.vertexCount*sizeof(int)); 
    //hipMalloc(&verticesRemaining, other.vertexCount*sizeof(int)); 
    //hipMalloc(&new_degrees_dev, other.vertexCount*sizeof(int)); 
}

__host__ __device__ Graph_GPU::~Graph_GPU(){
    //csr.~CSR_GPU();
    //hipFree(hasntBeenRemoved);
    //hipFree(verticesRemaining);
    //hipFree(new_degrees_dev);
}

