#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <cstdlib>
#include <iostream>
#include <string>
#include <sstream>      // std::stringstream

// We'll use a 3-tuple to store our 3d vector type
// rows, cols, vals
typedef thrust::tuple<int,int,int> Int3;

struct cmp : public std::binary_function<Int3,Int3,bool>
{
    __host__ __device__
        bool operator()(const Int3& a, const Int3& b) const
        {
            if (thrust::get<0>(a) != thrust::get<0>(b))
                return thrust::get<0>(a) < thrust::get<0>(b);
            else 
                return thrust::get<1>(a) < thrust::get<1>(b);
        }
};

int main(int argc, char *argv[])
{
    int N = atoi(argv[1]);
    int numEntries = atoi(argv[2]);

    std::cout << "building vecs" << std::endl;
    thrust::host_vector<int> col_vec(numEntries);
    thrust::host_vector<int> row_vec(numEntries);
    thrust::host_vector<int> val_vec(numEntries);

    thrust::host_vector<int> dimensions(numEntries);
    thrust::host_vector<int> ones(numEntries);


    // fill dimensions vector with Ns
    thrust::fill(dimensions.begin(), dimensions.end(), N);
    thrust::fill(ones.begin(), ones.end(), 1);

    std::cout << "generating vecs on host" << std::endl;

    thrust::generate(col_vec.begin(), col_vec.end(), rand);
    thrust::generate(row_vec.begin(), row_vec.end(), rand);
    thrust::generate(val_vec.begin(), val_vec.end(), rand);

    // compute Y = X mod N
    std::cout << "transforming vecs on host" << std::endl;

    thrust::transform(col_vec.begin(), col_vec.end(), dimensions.begin(), col_vec.begin(), thrust::modulus<int>());
    thrust::transform(row_vec.begin(), row_vec.end(), dimensions.begin(), row_vec.begin(), thrust::modulus<int>());
    thrust::transform(val_vec.begin(), val_vec.end(), dimensions.begin(), val_vec.begin(), thrust::modulus<int>());
    thrust::transform(val_vec.begin(), val_vec.end(), ones.begin(), val_vec.begin(), thrust::plus<int>());


    std::cout << "copying vecs from host to device" << std::endl;

    thrust::device_vector<int> col_vec_dev = col_vec;
    thrust::device_vector<int> row_vec_dev = row_vec;
    thrust::device_vector<int> val_vec_dev = val_vec;

    // METHOD #1
    // Defining a zip_iterator type can be a little cumbersome ...
    std::cout << "creating tuples" << std::endl;

    typedef thrust::device_vector<int>::iterator                     IntIterator;
    typedef thrust::tuple<IntIterator, IntIterator, IntIterator> IntIteratorTuple;
    typedef thrust::zip_iterator<IntIteratorTuple>                   Int3Iterator;

    std::cout << "creating iterators" << std::endl;

    // Now we'll create some zip_iterators for A and B
    Int3Iterator A_first = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev.begin(), col_vec_dev.begin(), val_vec_dev.begin()));
    Int3Iterator A_last  = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev.end(),   col_vec_dev.end(),   val_vec_dev.end()));
    //Int3Iterator B_first = thrust::make_zip_iterator(thrust::make_tuple(B0.begin(), B1.begin(), B2.begin()));
    std::cout << "sorting" << std::endl;

    thrust::sort(A_first, A_last, cmp());

    std::cout << "copying back to host vecs" << std::endl;

    col_vec = col_vec_dev;
    row_vec = row_vec_dev;
    val_vec = val_vec_dev;

    std::stringstream ss;
    std::string myMatrix;
    ss << "\t\tCOO Matrix" << std::endl;
    for (int i = 0; i<N; i++){
        ss << "\tcol " << i;
    }
    ss << std::endl;
    int row_index = 0;
    for (int i = 0; i < N; i++){
        ss << "row " << i;
        for( int j = 0; j < N; j++){
            if (row_vec[row_index] ==  i){
                if(j==col_vec[row_index]){
                    ss << "\t" << val_vec[row_index];
                    // Skip duplicate entries
                    while(row_vec[row_index] == i && j == col_vec[row_index]){
                        row_index++;
                    }
                } else {
                    ss << "\t" << 0;
                }
            } else {
                ss << "\t" << 0;
            }
        }        
        ss << std::endl;
    }
    ss << "Row indices" << std::endl;
    for(int i = 0; i< row_vec.size(); i++){
        ss << "\t" << row_vec[i];
    }
    ss << std::endl;
    ss << "Column indices" << std::endl;
    for(int i = 0; i< col_vec.size(); i++){
        ss << "\t" << col_vec[i];
    }
    ss << std::endl;
    ss << "values" << std::endl;
    for(int i = 0; i< val_vec.size(); i++){
        ss << "\t" << val_vec[i];
    }
    ss << std::endl;
    myMatrix = ss.str();

    std::cout << myMatrix << std::endl;
}