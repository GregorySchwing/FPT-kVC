#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <cstdlib>
#include <iostream>
#include <string>
#include <sstream>      // std::stringstream

// We'll use a 3-tuple to store our 3d vector type
// rows, cols, vals
typedef thrust::tuple<int,int,int> Int3;

struct cmp : public std::binary_function<Int3,Int3,bool>
{
    __host__ __device__
        bool operator()(const Int3& a, const Int3& b) const
        {
            if (thrust::get<0>(a) != thrust::get<0>(b))
                return thrust::get<0>(a) < thrust::get<0>(b);
            else 
                return thrust::get<1>(a) < thrust::get<1>(b);
        }
};

int main(int argc, char *argv[])
{
    int N = atoi(argv[1]);
    int numEntries = atoi(argv[2]);

/*
    int column_indices_a[numEntries], row_indices_a[numEntries], values_a[numEntries];
    int * column_indices_a_dev, * row_indices_a_dev, * values_a_dev;

    int column_indices_b[numEntries], row_indices_b[numEntries], values_b[numEntries];
    int * column_indices_b_dev, * row_indices_b_dev, * values_b_dev;

    hipMalloc( (void**)&column_indices_a_dev, numEntries * sizeof(int) );
    hipMalloc( (void**)&row_indices_a_dev, numEntries * sizeof(int) );
    hipMalloc( (void**)&values_a_dev, numEntries * sizeof(int) );

    hipMalloc( (void**)&column_indices_b_dev, numEntries * sizeof(int) );
    hipMalloc( (void**)&row_indices_b_dev, numEntries * sizeof(int) );
    hipMalloc( (void**)&values_b_dev, numEntries * sizeof(int) );

    int trialRow, trialCol;
    bool empty;
    for (int i = 0; i < numEntries; i++){
        do {
            empty = true;
            trialCol = std::rand() % N;
            trialRow = std::rand() %  N;
            for (int j = 0; j < i; j++){
                if (row_indices_a[j] == trialRow && column_indices_a[j] == trialCol)
                    empty = false;
            }
        } while (!empty);
        row_indices_a[i] = trialRow;
        column_indices_a[i] = trialCol;
        values_a[i] =  std::rand() % N + 1;
    }

    for (int i = 0; i < numEntries; i++){
        do {
            empty = true;
            trialCol = std::rand() % N;
            trialRow = std::rand() %  N;
            for (int j = 0; j < i; j++){
                if (row_indices_b[j] == trialRow && column_indices_b[j] == trialCol)
                    empty = false;
            }
        } while (!empty);
        row_indices_b[i] = trialRow;
        column_indices_b[i] = trialCol;
        values_b[i] =  std::rand() % N + 1;
    }

    hipMemcpy(column_indices_a_dev, column_indices_a, numEntries * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_indices_a_dev, row_indices_a, numEntries * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(values_a_dev, values_a, numEntries * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(column_indices_b_dev, column_indices_b, numEntries * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_indices_b_dev, row_indices_b, numEntries * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(values_b_dev, values_b, numEntries * sizeof(int), hipMemcpyHostToDevice);
*/
    // generate 16M random numbers on the host

    std::cout << "building vecs" << std::endl;
    thrust::device_vector<int> col_vec(numEntries);
    thrust::device_vector<int> row_vec(numEntries);
    thrust::device_vector<int> val_vec(numEntries);

    thrust::device_vector<int> dimensions(numEntries);
    thrust::device_vector<int> ones(numEntries);


    // fill dimensions with twos
    thrust::fill(dimensions.begin(), dimensions.end(), N);
    thrust::fill(ones.begin(), ones.end(), 1);

    std::cout << "generating vecs on gpu" << std::endl;

    thrust::generate(col_vec.begin(), col_vec.end(), rand);
    thrust::generate(row_vec.begin(), row_vec.end(), rand);
    thrust::generate(val_vec.begin(), val_vec.end(), rand);

    // compute Y = X mod 2
    std::cout << "transforming vecs on gpu" << std::endl;

    thrust::transform(col_vec.begin(), col_vec.end(), dimensions.begin(), col_vec.begin(), thrust::modulus<int>());
    thrust::transform(row_vec.begin(), row_vec.end(), dimensions.begin(), row_vec.begin(), thrust::modulus<int>());
    thrust::transform(val_vec.begin(), val_vec.end(), dimensions.begin(), val_vec.begin(), thrust::modulus<int>());
    thrust::transform(val_vec.begin(), val_vec.end(), ones.begin(), val_vec.begin(), thrust::plus<int>());

    // METHOD #1
    // Defining a zip_iterator type can be a little cumbersome ...
    std::cout << "creating tuples" << std::endl;

    typedef thrust::device_vector<int>::iterator                     IntIterator;
    typedef thrust::tuple<IntIterator, IntIterator, IntIterator> IntIteratorTuple;
    typedef thrust::zip_iterator<IntIteratorTuple>                   Int3Iterator;

    std::cout << "creating iterators" << std::endl;

    // Now we'll create some zip_iterators for A and B
    Int3Iterator A_first = thrust::make_zip_iterator(thrust::make_tuple(col_vec.begin(), row_vec.begin(), val_vec.begin()));
    Int3Iterator A_last  = thrust::make_zip_iterator(thrust::make_tuple(col_vec.end(),   row_vec.end(),   val_vec.end()));
    //Int3Iterator B_first = thrust::make_zip_iterator(thrust::make_tuple(B0.begin(), B1.begin(), B2.begin()));
        std::cout << "sorting" << std::endl;

    thrust::sort(A_first, A_last, cmp());

    std::cout << "creating host vecs" << std::endl;

    thrust::host_vector<int> col_vec_host(numEntries);
    thrust::host_vector<int> row_vec_host(numEntries);
    thrust::host_vector<int> val_vec_host(numEntries);

    col_vec_host = col_vec;
    row_vec_host = row_vec;
    val_vec_host = val_vec;

    std::stringstream ss;
    std::string myMatrix;
    ss << "\t\tCOO Matrix" << std::endl;
    for (int i = 0; i<numEntries; i++){
        ss << "\tcol " << i;
    }
    ss << std::endl;
    int row_index = 0;
    for (int i = 0; i < numEntries; i++){
        ss << "row " << i;
        for( int j = 0; j < numEntries; j++){
            if (row_vec_host[row_index] ==  i){
                if(j==col_vec_host[row_index]){
                    ss << "\t" << val_vec_host[row_index];
                    // Skip duplicate entries
                    while(row_vec_host[row_index] == i && j == col_vec_host[row_index]){
                        row_index++;
                    }
                } else {
                    ss << "\t" << 0;
                }
            } else {
                ss << "\t" << 0;
            }
        }        
        ss << std::endl;
    }
    ss << "Row indices" << std::endl;
    for(int i = 0; i< row_vec_host.size(); i++){
        ss << "\t" << row_vec_host[i];
    }
    ss << std::endl;
    ss << "Column indices" << std::endl;
    for(int i = 0; i< col_vec_host.size(); i++){
        ss << "\t" << col_vec_host[i];
    }
    ss << std::endl;
    ss << "values" << std::endl;
    for(int i = 0; i< val_vec_host.size(); i++){
        ss << "\t" << val_vec_host[i];
    }
    ss << std::endl;
    myMatrix = ss.str();

    std::cout << myMatrix << std::endl;
}